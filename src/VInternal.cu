#include "hip/hip_runtime.h"
/************************************************************************\

  Copyright 1997 The University of North Carolina at Chapel Hill.
  All Rights Reserved.

  Permission to use, copy, modify and distribute this software
  and its documentation for educational, research and non-profit
  purposes, without fee, and without a written agreement is
  hereby granted, provided that the above copyright notice and
  the following three paragraphs appear in all copies.

  IN NO EVENT SHALL THE UNIVERSITY OF NORTH CAROLINA AT CHAPEL
  HILL BE LIABLE TO ANY PARTY FOR DIRECT, INDIRECT, SPECIAL,
  INCIDENTAL, OR CONSEQUENTIAL DAMAGES, INCLUDING LOST PROFITS,
  ARISING OUT OF THE USE OF THIS SOFTWARE AND ITS DOCUMENTATION,
  EVEN IF THE UNIVERSITY OF NORTH CAROLINA HAVE BEEN ADVISED OF
  THE POSSIBILITY OF SUCH DAMAGES.


  Permission to use, copy, modify and distribute this software
  and its documentation for educational, research and non-profit
  purposes, without fee, and without a written agreement is
  hereby granted, provided that the above copyright notice and
  the following three paragraphs appear in all copies.

  THE UNIVERSITY OF NORTH CAROLINA SPECIFICALLY DISCLAIM ANY
  WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
  WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
  PURPOSE.  THE SOFTWARE PROVIDED HEREUNDER IS ON AN "AS IS"
  BASIS, AND THE UNIVERSITY OF NORTH CAROLINA HAS NO OBLIGATION
  TO PROVIDE MAINTENANCE, SUPPORT, UPDATES, ENHANCEMENTS, OR
  MODIFICATIONS.


   --------------------------------- 
  |Please send all BUG REPORTS to:  |
  |                                 |
  |   geom@cs.unc.edu               |
  |                                 |
   ---------------------------------
  
     
  The authors may be contacted via:

  US Mail:  A. Pattekar/J. Cohen/T. Hudson/S. Gottschalk/M. Lin/D. Manocha
            Department of Computer Science
            Sitterson Hall, CB #3175
            University of N. Carolina
            Chapel Hill, NC 27599-3175
	    
  Phone:    (919)962-1749
	    
  EMail:    geom@cs.unc.edu

\************************************************************************/

/************************************************************************\
Filename: VInternal.C
--
Description: This file implements the member functions of the class vinternal.c

\************************************************************************/



#include <iostream>
#include <string.h>     //for memset and memcpy.
#include "VInternal.H"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "CycleTimer.h"
#include "objects.h"
#include "NBody.H"
#include <math.h>  

inline double GT(double a, double b)
{
  return (( (a) > (b) ) ? (a) : (b));
}

void add_overlap_pair(int id1, int id2, NBody * obj) //add a pair to the set of
  {                                     //overlapping pairs.
      if (id1 != id2)
	        obj->overlapping_pairs.AddPair(id1, id2);
  }
  
  void del_overlap_pair(int id1, int id2, NBody * obj) //delete a pair from the set.
    {
      if (id1 != id2)
	        obj->overlapping_pairs.DelPair(id1, id2);
    }

int overlaps(AABB *obj1, AABB *obj2) //to check it the two AABBs overlap.
{
  int coord;
  for (coord=0; coord<3; coord++)
    {
      if (obj1->lo->val[coord] < obj2->lo->val[coord])
	{
	  if (obj2->lo->val[coord] > obj1->hi->val[coord])
	    return 0;
	}
      else
	{
	  if (obj1->lo->val[coord] > obj2->hi->val[coord])
	    return 0;
	}
    }
  
  return 1;
}

void NBody_constructor(NBody *obj, int mySize)  //constructor.
{
  for (int i =0; i < 3;i++){
    obj->head[i] = new EndPoint;
    obj->head[i]->minmax = MIN;
    obj->head[i]->val[0] = - (1<<29);
    obj->head[i]->val[1] = - (1<<29);
    obj->head[i]->val[2] = - (1<<29);
  }

 
  obj->size = mySize;
  obj->AABB_arr = new AABB*[obj->size];  //allocate the dynamic array and initialize
  int i;
  for (i=0; i<obj->size; i++)      //all its elements to NULL.
    obj->AABB_arr[i] = NULL;
  
}

void add_node(EndPoint* node, int dim, EndPoint* prevNode ) {
        EndPoint* temp = prevNode->next[dim];
        node->next[dim] = temp;
        node->prev[dim] = prevNode;
        prevNode->next[dim] = node;
        if (temp != NULL)
          temp->prev[dim] = node;
    }
    
void delete_node(EndPoint* delnode, int dim) {
        EndPoint* delprev = delnode->prev[dim];
        EndPoint* delnext = delnode->next[dim];
        if (delprev !=NULL)
          delprev->next[dim] = delnext;
        if (delnext != NULL)
          delnext->prev[dim] = delprev;
    }


void updatetempTrans(int id, double trans[][4], NBody * obj){
  
  AABB *current = obj->AABB_arr[id];
  

  double new_center[3], min[3], max[3];
  AABB dummy;       //we need these so that we can use the same function
  EndPoint lo = (EndPoint){.minmax = MIN, .aabb = &dummy};
  EndPoint hi = (EndPoint){.minmax = MAX, .aabb = &dummy};
  dummy.lo = &lo;
  dummy.hi = &hi;


  for (int dim = 0; dim < 3; dim ++){
    new_center[dim] = current->center[0] * trans[dim][0] + current->center[1] * trans[dim][1] + current->center[2] * trans[dim][2] + trans[dim][3];
    min[dim] = lo.val[dim] = new_center[dim] - current->radius;
    max[dim] =  hi.val[dim] = new_center[dim] + current->radius;
    
  }
  
  //update all the three lists by moving the endpoint to correct position.
  int coord;
  for (coord=0; coord<3; coord++)
    {
      int direction;
      EndPoint *temp;
      
      //set the direction of motion of the endpoint along the list.
      if (current->lo->val[coord] > min[coord])
	        direction = REVERSE;
      else if (current->lo->val[coord] <min[coord])
	        direction = FORWARD;
      else
	        direction = NOCHANGE;
      
  if (direction == REVERSE) //backward motion....
	{

	  temp = current->lo;
	  while ((temp != NULL) && (temp->val[coord] > min[coord]))
		{
		  if (temp->minmax == MAX){
		    if (overlaps(temp->aabb, &dummy))
		      add_overlap_pair(temp->aabb->id, current->id, obj);
      } 
		  temp = temp->prev[coord];
		}

    delete_node(current->lo,coord); 
    add_node(current->lo, coord, temp);

	  current->lo->val[coord] = min[coord];
	  
	  //then update the "hi" endpoint of the interval.
	  if (current->hi->val[coord] != max[coord])
	    {
	      temp = current->hi;
	      
	  while (temp->val[coord] > max[coord])
		{
		if ( (temp->minmax == MIN) && (overlaps(temp->aabb, current)) )
		    del_overlap_pair(temp->aabb->id, current->id, obj);
		  temp = temp->prev[coord];
		}
	  
    delete_node(current->hi, coord);
    add_node(current->hi, coord, temp);
	  current->hi->val[coord] = max[coord];
	    }
	}
  else if (direction == FORWARD) //forward motion....
	{
	  //here, we first update the "hi" endpoint.
	  if (current->hi->next[coord] != NULL)
	    {
	      temp = current->hi;
	      while ( (temp->next[coord] != NULL) && (temp->next[coord]->val[coord]< max[coord]) )
		{
		  if (temp->minmax == MIN)
		    if (overlaps(temp->aabb, &dummy))
		      add_overlap_pair(temp->aabb->id, current->id, obj);
		  temp = temp->next[coord];
		}
	  delete_node(current->hi, coord);
    add_node(current->hi, coord, temp);

	    }
	  current->hi->val[coord] = max[coord];
	  
	  //then, update the "lo" endpoint of the interval.
	  temp = current->lo;
	  
	  while (temp->val[coord] < min[coord])
	    {
	    if ( (temp->minmax == MAX) && (overlaps(temp->aabb, current)) )
		      del_overlap_pair(temp->aabb->id, current->id, obj);
	      
	      temp = temp->next[coord];
	    }
	  delete_node(current->lo, coord);
    add_node(current->lo, coord, temp->prev[coord]);
	  current->lo->val[coord] = min[coord];
	}   
    }
}



double findRadius(AABB *curr, Object *b){
  double val = 0.0;

  for (int i=0; i<(b->num_tris); i++)
    {
      double cur_rad1_sq = 0;
      double cur_rad2_sq = 0;
      double cur_rad3_sq = 0;
      for (int w=0; w<3; w++)
      {
        double my_num1 = curr->center[w] - b->tris[i].p1[w];
        double my_num2 = curr->center[w] - b->tris[i].p2[w];
        double my_num3 = curr->center[w] - b->tris[i].p3[w];
        cur_rad1_sq += pow(my_num1, 2);
        cur_rad2_sq += pow(my_num2, 2);
        cur_rad3_sq += pow(my_num3, 2);
      }
               
      double max_rad_sq = GT(cur_rad1_sq, GT(cur_rad2_sq,cur_rad3_sq));
      
      val = GT(max_rad_sq, val);
    }
    return sqrt(val) * 1.0001;
}

void findCenter(AABB *curr, Object *b){
  curr->center[0] = curr->center[1] = curr->center[2] = 0.0;
  
  for (int dim = 0; dim < 3; dim++){
  for (int i=0; i<(b->num_tris); i++)
      curr->center[dim] += b->tris[i].p1[dim] + b->tris[i].p2[dim] + b->tris[i].p3[dim];
  curr->center[dim] /= (3*b->num_tris); 
  }
}


void AddObject(int id, Object *b, NBody * obj) //add a new object
{
  AABB *curr = new AABB;
  
  curr->id = id; //set the id to the given value.
  findRadius(curr, b);  
  curr->radius = findRadius(curr, b);
  EndPoint lo = (EndPoint){.minmax = MIN, .aabb = curr};
  EndPoint hi = (EndPoint){.minmax = MAX, .aabb = curr};
  curr->lo = &lo;
  curr->hi = &hi;


  for (int w=0; w<3; w++){
  curr->lo->val[w] = curr->center[w] - curr->radius; 
  curr->hi->val[w] = curr->center[w] + curr->radius;
  }


  for (int i=0; i<obj->size; i++)      //Now, check the overlap of this AABB with 
  {                         //with all other AABBs and add the pair to
  if (obj->AABB_arr[i])        //the set of overlapping pairs if reqd.
	    if (overlaps(curr, obj->AABB_arr[i]))
	        add_overlap_pair(curr->id, i, obj);
    }

    std::cout << obj->size << "\n";
     std::cout << id << "\n";

  obj->AABB_arr[id] = curr;  //finally, insert the AABB in AABB_arr.
  

  //Now, for each of the three co-ordinates, insert the interval
  //in the correspoding list. 
  int coord;
  for (coord=0; coord <3; coord++)
    {
      EndPoint *current = obj->head[coord];
    
	  while ( current->next[coord] && (current->next[coord]->val[coord] < curr->hi->val[coord]) )
	    current = current->next[coord];
    add_node(curr->hi, coord, current);

      //now, insert the "lo" endpoint.
    current = obj->head[coord];

    while ( (current->next[coord] != NULL) && (current->val[coord] < curr->lo->val[coord]) )
	    current = current->next[coord];

    add_node(curr->lo, coord, current);
      
    }
  
}



void deleteObjects(int id, NBody * obj) //deleting an AABB with given id.
{

  AABB *curr = obj->AABB_arr[id];  //this is the AABB to be deleted.
  obj->AABB_arr[id] = NULL;        //remove it from the AABB array.
  
  //first, we delete all the three intervals from the corresponding lists.
  int coord;
  for (coord=0; coord<3; coord++)
    {
      //first delete the "lo" endpoint of the interval.
      if (curr->lo->prev[coord] == NULL)
	obj->head[coord] = curr->lo->next[coord];
      else
	curr->lo->prev[coord]->next[coord] = curr->lo->next[coord];
      
      curr->lo->next[coord]->prev[coord] = curr->lo->prev[coord];
      
      //then, delete the "hi" endpoint.
      if (curr->hi->prev[coord] == NULL)
	obj->head[coord] = curr->hi->next[coord];
      else
	curr->hi->prev[coord]->next[coord] = curr->hi->next[coord];
      
      if (curr->hi->next[coord] != NULL)
	curr->hi->next[coord]->prev[coord] = curr->hi->prev[coord];
      
    }
  
  //delete all entries involving this id from the set of 
  //overlapping pairs.
  obj->overlapping_pairs.DelPairsInvolvingId(id);
  
  //de-allocate the memory
  delete curr->lo;
  delete curr->hi;
  delete curr;
}


VCInternal::VCInternal(int mySize)
{
  state = VCstate_default;
  next_id = 0;
  
  vc_objects = new VCObject*[mySize]; //allocate the array.
  NBody_constructor(&nbody, mySize);
  int i;
  for (i=0; i<mySize; i++)
    vc_objects[i] = NULL;
  
  disabled.Clear();  //to begin with, no pairs are disabled.
}


VCInternal::~VCInternal()
{

  //deallocate the memory.
  int i;
  for (i=0; i<size; i++)
    {
      if (vc_objects[i])
	{
	  delete vc_objects[i]->b;
	  delete vc_objects[i];
	}
    }
  delete [] vc_objects;
}

//1. check if the size fit in
//2 assign the object an id and activate the object
int VCInternal::NewObject(int *id) //create a new object in the database.
{

  //increase the size of the "vc_objects" array if required.
  if (next_id >= size) 
    {
      int newsize = (next_id >= 2*size) ? (next_id+1) : 2*size;
      VCObject **temp = new VCObject*[newsize];
      int i;
      for (i=0; i<size; i++)
	temp[i] = vc_objects[i];
      for (i=size; i<newsize; i++)
	temp[i] = NULL;
      delete [] vc_objects;
      vc_objects = temp;
      size = newsize;
      
    }
  
  //allocate a new object.
  vc_objects[next_id] = new VCObject;
  
  *id = next_id;  //for returning the id generated by VCollide.
  current_id = next_id;
  vc_objects[next_id]->id = next_id;
  vc_objects[next_id]->b = new Object;
  vc_objects[next_id]->b->BeginModel();
  //_state = 1;//default the object is activate
  next_id++; 
  
  return 0;
}

int VCInternal::AddTri(double v1[], double v2[], double v3[]) 
{                     

  vc_objects[current_id]->b->AddTri(v1, v2, v3, 0);  //add triangle.
  return 0;
}

// 1. add current object to n body
// 2. have RAPID build the OBB tree.
// 3. initialize trans
int VCInternal::EndObject(void)
{  

  AddObject(current_id, vc_objects[current_id]->b, &nbody);
  
  vc_objects[current_id]->b->EndModel();

  memset( ( (void *)vc_objects[current_id]->trans), 0, 16*sizeof(double) );
  vc_objects[current_id]->trans[0][0] = 1.0;
  vc_objects[current_id]->trans[1][1] = 1.0;
  vc_objects[current_id]->trans[2][2] = 1.0;
  vc_objects[current_id]->trans[3][3] = 1.0;
  
  return 0;
  
}


int VCInternal::UpdateTrans(int id, double t[][4])
{           

  VCObject *current = vc_objects[id];
  
  //update the private copy of the transformation matrix.
  memcpy((void *)current->trans, (void *)t, 16*sizeof(double));
  
  //have the nbody database update itself appropriately.
  //updateteTrans(current->id, t, &nbody);
  updatetempTrans(current->id, t, &nbody);
  

  return 0;
  
}

int VCInternal::ActivatePair(int id1, int id2)
{
      disabled.DelPair(id1, id2);
      return 0;
}

int VCInternal::DeactivatePair(int id1, int id2)
{

      if (id1!=id2)
	disabled.AddPair(id1, id2);
      
    return 0;
}

int VCInternal::DeleteObject(int id) //delete an object from the database.
{


      delete vc_objects[id]->b; //delete the RAPID box.
      delete vc_objects[id];    //delete the object.
      vc_objects[id] = NULL; 
      
      disabled.DelPairsInvolvingId(id);

      deleteObjects(id, &nbody); //delete the object from the nbody database.
      return 0;

  
}

int VCInternal::Collide(void)  //perform collision detection.
{

  
  //Clear the results from earlier collision tests.
  report_data.Clear();
  
  //Simultaneously traverse the "overlapping_pairs" database and the 
  //"disabled_pairs" database and make calls to the RAPID collision
  //detection routine where required.
  int i;
   //std::cout<< nbody.overlapping_pairs.size<< std::endl;
  for (i=0; i<nbody.overlapping_pairs.size; i++)
    {
      Elem *curr_ovrlp = nbody.overlapping_pairs.arr[i];
      
      Elem *curr_disabled = i<disabled.size ? disabled.arr[i]: NULL;
      
      
	  while (curr_ovrlp != NULL)
		{

		  while (curr_disabled && curr_disabled->id <= curr_ovrlp->id)
			      curr_disabled = curr_disabled->next;

      double R1[3][3], T1[3], R2[3][3], T2[3];
      for (int index = 0; index < 9; index++){
        int x = index/3;
        int y = index%3;
        R1[x][y] = vc_objects[i]->trans[x][y];
        R2[x][y] = vc_objects[curr_ovrlp->id]->trans[x][y];
      }

      for (int index = 0; index < 3; index++){
        T1[index] = vc_objects[i]->trans[index][3];
        T2[index] = vc_objects[curr_ovrlp->id]->trans[index][3];
      }

			      //call the RAPID collision detection routine.
			::Collide(R1, T1, vc_objects[i]->b, R2, T2, vc_objects[curr_ovrlp->id]->b, FIRST_CONTACT);
			      
			      //if there is a collision, then add the pair to the
			      //collision report database.
			if (Object_num_contacts != 0)
			      report_data.AddPair(i, vc_objects[curr_ovrlp->id]->id);
			      
		  curr_ovrlp = curr_ovrlp->next;
		}
    }
  return 0;
}


//report the results of collision detection.
//sz is the size of the array pointed to by vcrep. If sz is less than
//the number of collision pairs, then fill the array with first sz number
//of collision pairs.
//Returns the total number of collision pairs.
int VCInternal::Report(int sz, VCReportType *vcrep)
{
  int no_of_colliding_pairs=0;
  int vc_rep_count = 0;
  
  int i;
  for (i=0; i<report_data.size; i++)
    {
      Elem *current;
      for (current=report_data.arr[i]; current != NULL; current=current->next)
	{
	  no_of_colliding_pairs++;
	  if (vc_rep_count <sz) //if the array is not full yet, then 
	    {                   //fill the data in it.
	      vcrep[vc_rep_count].id1 = i;
	      vcrep[vc_rep_count].id2 = current->id;
	      vc_rep_count++;
	    }
	}
      
    }
  return no_of_colliding_pairs;
}


