#include "hip/hip_runtime.h"
/************************************************************************\

  Copyright 1997 The University of North Carolina at Chapel Hill.
  All Rights Reserved.

  Permission to use, copy, modify and distribute this software
  and its documentation for educational, research and non-profit
  purposes, without fee, and without a written agreement is
  hereby granted, provided that the above copyright notice and
  the following three paragraphs appear in all copies.

  IN NO EVENT SHALL THE UNIVERSITY OF NORTH CAROLINA AT CHAPEL
  HILL BE LIABLE TO ANY PARTY FOR DIRECT, INDIRECT, SPECIAL,
  INCIDENTAL, OR CONSEQUENTIAL DAMAGES, INCLUDING LOST PROFITS,
  ARISING OUT OF THE USE OF THIS SOFTWARE AND ITS DOCUMENTATION,
  EVEN IF THE UNIVERSITY OF NORTH CAROLINA HAVE BEEN ADVISED OF
  THE POSSIBILITY OF SUCH DAMAGES.


  Permission to use, copy, modify and distribute this software
  and its documentation for educational, research and non-profit
  purposes, without fee, and without a written agreement is
  hereby granted, provided that the above copyright notice and
  the following three paragraphs appear in all copies.

  THE UNIVERSITY OF NORTH CAROLINA SPECIFICALLY DISCLAIM ANY
  WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
  WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
  PURPOSE.  THE SOFTWARE PROVIDED HEREUNDER IS ON AN "AS IS"
  BASIS, AND THE UNIVERSITY OF NORTH CAROLINA HAS NO OBLIGATION
  TO PROVIDE MAINTENANCE, SUPPORT, UPDATES, ENHANCEMENTS, OR
  MODIFICATIONS.


   --------------------------------- 
  |Please send all BUG REPORTS to:  |
  |                                 |
  |   geom@cs.unc.edu               |
  |                                 |
   ---------------------------------
  
     
  The authors may be contacted via:

  US Mail:  A. Pattekar/J. Cohen/T. Hudson/S. Gottschalk/M. Lin/D. Manocha
            Department of Computer Science
            Sitterson Hall, CB #3175
            University of N. Carolina
            Chapel Hill, NC 27599-3175
	    
  Phone:    (919)962-1749
	    
  EMail:    geom@cs.unc.edu

\************************************************************************/

/************************************************************************\
Filename: VInternal.C
--
Description: This file implements the member functions of the class vinternal.c

\************************************************************************/



#include <iostream>
#include <string.h>     //for memset and memcpy.
#include "VInternal.H"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "CycleTimer.h"
#include "objects.h"
#include "NBody.H"
#include <math.h>  
const int DEFAULT_SIZE=10; //some arbitrary default size for "vc_objects" array.

  void add_overlap_pair(int id1, int id2, NBody * obj) //add a pair to the set of
    {                                     //overlapping pairs.
      if (id1 != id2)
	obj->overlapping_pairs.AddPair(id1, id2);
    }
  
  void del_overlap_pair(int id1, int id2, NBody * obj) //delete a pair from the set.
    {
      if (id1 != id2)
	obj->overlapping_pairs.DelPair(id1, id2);
    }

int overlaps(AABB *obj1, AABB *obj2) //to check it the two AABBs overlap.
{
  int coord;
  for (coord=0; coord<3; coord++)
    {
      if (obj1->lo->val[coord] < obj2->lo->val[coord])
	{
	  if (obj2->lo->val[coord] > obj1->hi->val[coord])
	    return 0;
	}
      else
	{
	  if (obj1->lo->val[coord] > obj2->hi->val[coord])
	    return 0;
	}
    }
  
  return 1;
}
void add_node(EndPoint* node, int dim, EndPoint* prevNode ) {
        EndPoint* temp = prevNode->next[dim];
        node->next[dim] = temp;
        node->prev[dim] = prevNode;
        prevNode->next[dim] = node;
        if (temp != NULL)
          temp->prev[dim] = node;
    }
    
void delete_node(EndPoint* delnode, int dim) {
        EndPoint* delprev = delnode->prev[dim];
        EndPoint* delnext = delnode->next[dim];
        if (delprev !=NULL)
          delprev->next[dim] = delnext;
        if (delnext != NULL)
          delnext->prev[dim] = delprev;
    }
void updatetempTrans(int id, double trans[][4], NBody * obj){
  
  AABB *current = obj->AABB_arr[id]; //the given object exists !
  
  //compute the new position of the AABB center.
  double new_center[3], min[3], max[3];
  AABB dummy;       //we need these so that we can use the same function
  EndPoint lo, hi; 
  dummy.lo = &lo;
  dummy.hi = &hi;
  lo.minmax = MIN;
  hi.minmax = MAX;
  lo.aabb = &dummy;
  hi.aabb = &dummy;

  for (int dim = 0; dim < 3; dim ++){
    new_center[dim] = current->center[0] * trans[dim][0] + current->center[1] * trans[dim][1] + current->center[2] * trans[dim][2] + trans[dim][3];
    min[dim] = lo.val[dim] = new_center[dim] - current->radius;
    max[dim] =  hi.val[dim] = new_center[dim] + current->radius;
    
  }
  
  //update all the three lists by moving the endpoint to correct position.
  int coord;
  for (coord=0; coord<3; coord++)
    {
      int direction;
      EndPoint *temp;
      
      //set the direction of motion of the endpoint along the list.
      if (current->lo->val[coord] > min[coord])
	        direction = REVERSE;
      else if (current->lo->val[coord] <min[coord])
	        direction = FORWARD;
      else
	        direction = NOCHANGE;
      
      if (direction == REVERSE) //backward motion....
	{
	  //first update the "lo" endpoint of the interval
	  if (current->lo->prev[coord] != NULL)
	    {
	      temp = current->lo;
	      while ((temp != NULL) && (temp->val[coord] > min[coord]))
		{
		  if (temp->minmax == MAX){
		    if (overlaps(temp->aabb, &dummy))
		      add_overlap_pair(temp->aabb->id, current->id, obj);
      } 
		  temp = temp->prev[coord];
		}

    delete_node(current->lo,coord); 

  // if it is the smallest
	  if (temp == NULL)
		{
		  current->lo->prev[coord] = NULL;
		  current->lo->next[coord] = obj->elist[coord];
		  obj->elist[coord]->prev[coord] = current->lo;
		  obj->elist[coord] = current->lo;
		}
	  else
		{
      add_node(current->lo, coord, temp);
		} 
	    }
	  
	  current->lo->val[coord] = min[coord];
	  
	  //then update the "hi" endpoint of the interval.
	  if (current->hi->val[coord] != max[coord])
	    {
	      temp = current->hi;
	      
	  while (temp->val[coord] > max[coord])
		{
		if ( (temp->minmax == MIN) && (overlaps(temp->aabb, current)) )
		    del_overlap_pair(temp->aabb->id, current->id, obj);
		  temp = temp->prev[coord];
		}
	  
    delete_node(current->hi, coord);
    add_node(current->hi, coord, temp);

	      current->hi->val[coord] = max[coord];
	    }
	}
      else if (direction == FORWARD) //forward motion....
	{
	  //here, we first update the "hi" endpoint.
	  if (current->hi->next[coord] != NULL)
	    {
	      temp = current->hi;
	      while ( (temp->next[coord] != NULL) && (temp->val[coord] < max[coord]) )
		{
		  if (temp->minmax == MIN)
		    if (overlaps(temp->aabb, &dummy))
		      add_overlap_pair(temp->aabb->id, current->id, obj);
		  
		  temp = temp->next[coord];
		}
	  
	  if (temp->val[coord] < max[coord])
		{
      delete_node(current->hi, coord);
		  current->hi->prev[coord] = temp;
		  current->hi->next[coord] = NULL;
		  temp->next[coord] = current->hi;
		}
	  else if (current->hi->val[coord] != max[coord])
		{
      delete_node(current->hi, coord);
      add_node(current->hi, coord, temp->prev[coord]);
		}
	    }
	  current->hi->val[coord] = max[coord];
	  
	  //then, update the "lo" endpoint of the interval.
	  temp = current->lo;
	  
	  while (temp->val[coord] < min[coord])
	    {
	      if ( (temp->minmax == MAX) && (overlaps(temp->aabb, current)) )
		del_overlap_pair(temp->aabb->id, current->id, obj);
	      
	      temp = temp->next[coord];
	    }
	  
	  if (current->lo->prev[coord] != NULL)
	    current->lo->prev[coord]->next[coord] = current->lo->next[coord];
	  else
	    obj->elist[coord] = current->lo->next[coord];
	  current->lo->next[coord]->prev[coord] = current->lo->prev[coord];
	  current->lo->prev[coord] = temp->prev[coord];
	  current->lo->next[coord] = temp;
	  if (temp->prev[coord] != NULL)
	    temp->prev[coord]->next[coord] = current->lo;
	  else
	    obj->elist[coord] = current->lo;
	  temp->prev[coord] = current->lo;
	  current->lo->val[coord] = min[coord];
	}   
    }
}

inline double GT(double a, double b)
{
  return (( (a) > (b) ) ? (a) : (b));
}

void AddObject(int id, Object *b, NBody * obj) //add a new object
{
  AABB *curr = new AABB;
  
  curr->id = id; //set the id to the given value.
  
  //The centroid of the object is computed and this is taken to be the
  //center of the AABB. 找到AABB的中心 
  curr->center[0] = curr->center[1] = curr->center[2] = 0.0;
  
  int i;
  for (i=0; i<(b->num_tris); i++)
    {
      curr->center[0] += b->tris[i].p1[0] + b->tris[i].p2[0] + b->tris[i].p3[0];
      curr->center[1] += b->tris[i].p1[1] + b->tris[i].p2[1] + b->tris[i].p3[1];
      curr->center[2] += b->tris[i].p1[2] + b->tris[i].p2[2] + b->tris[i].p3[2];
    }
  
  curr->center[0] /= (3*b->num_tris); 
  curr->center[1] /= (3*b->num_tris);
  curr->center[2] /= (3*b->num_tris);
  //------------------
  

  //The "radius" of the AABB is computed as the maximum distance of the AABB
  //center from any of the vertices of the object.
  curr->radius = 0.0;

  for (i=0; i<(b->num_tris); i++)
    {
      double cur_rad1_sq = 0;
      double cur_rad2_sq = 0;
      double cur_rad3_sq = 0;
      for (int w=0; w<3; w++)
      {
        double my_num1 = curr->center[w] - b->tris[i].p1[w];
        double my_num2 = curr->center[w] - b->tris[i].p2[w];
        double my_num3 = curr->center[w] - b->tris[i].p3[w];
        cur_rad1_sq += pow(my_num1, 2);
        cur_rad2_sq += pow(my_num2, 2);
        cur_rad3_sq += pow(my_num3, 2);
      }
               
      
      double max_rad_sq = GT(cur_rad1_sq, GT(cur_rad2_sq,cur_rad3_sq));
      
      curr->radius = GT(max_rad_sq, curr->radius);
      
    }

  curr->radius = sqrt(curr->radius);
  curr->radius *= 1.0001;  //add a 0.01% buffer.
  curr->lo = new EndPoint;
  curr->hi = new EndPoint;
  curr->lo->minmax = MIN;
  curr->hi->minmax = MAX;
  curr->lo->aabb = curr;
  curr->hi->aabb = curr;
  double min[3], max[3];
  
  for (int w=0; w<3; w++){
  min[w] = curr->center[w] - curr->radius; 
  max[w] = curr->center[w] + curr->radius;
  curr->lo->val[w] = min[w];
  curr->hi->val[w] = max[w];
  }


  for (i=0; i<obj->size; i++)      //Now, check the overlap of this AABB with 
    {                         //with all other AABBs and add the pair to
      if (obj->AABB_arr[i])        //the set of overlapping pairs if reqd.
	if (overlaps(curr, obj->AABB_arr[i]))
	  add_overlap_pair(curr->id, i, obj);
    }

  if (id >= obj->size)    //increase the size of the dynamic array if necessary.
      {
	int newsize = (id >= 2*obj->size) ? (id+1) : 2*obj->size;

	AABB **temp = new AABB*[newsize];
	int i;
	for (i=0; i<obj->size; i++)
	  temp[i] = obj->AABB_arr[i];
	for (i=obj->size; i<newsize; i++)
	  temp[i] = NULL;
	delete [] obj->AABB_arr;
	obj->AABB_arr = temp;
	obj->size = newsize;
      }
  
  obj->AABB_arr[id] = curr;  //finally, insert the AABB in AABB_arr.
  

  //Now, for each of the three co-ordinates, insert the interval
  //in the correspoding list. 
  int coord;
  for (coord=0; coord <3; coord++)
    {
      EndPoint *current = obj->elist[coord];
      
      //first insert the "hi" endpoint.
      if (current == NULL)    //if the list is empty, insert in front.
	{
	 obj->elist[coord] = curr->hi;
	  curr->hi->prev[coord] = curr->hi->next[coord] = NULL;
	}
      else  //otherwise, find the correct location in the list and
	{   //insert there. Note: the list is sorted.
	  while ( (current->next[coord] != NULL) && (current->val[coord] < curr->hi->val[coord]) )
	    current = current->next[coord];
	  
	  
	  if (current->val[coord] >= curr->hi->val[coord])
	    {
	      curr->hi->prev[coord] = current->prev[coord];
	      curr->hi->next[coord] = current;
	      if (current->prev[coord] == NULL)
		obj->elist[coord] = curr->hi;
	      else
		current->prev[coord]->next[coord] = curr->hi;
	      
	      current->prev[coord] = curr->hi;
	    }
	  else
	    {
	      curr->hi->prev[coord] = current;
	      curr->hi->next[coord] = NULL;
	      current->next[coord] = curr->hi;
	    }
	}
      
      //now, insert the "lo" endpoint.
      current = obj->elist[coord];
      
      //at this point, the list cannot be empty since we have already 
      //inserted the "hi" endpoint. So, we straightaway look for the 
      //correct location in the non-empty list and insert at that location.
      while ( (current->next[coord] != NULL) && (current->val[coord] < curr->lo->val[coord]) )
	current = current->next[coord];
      
      if (current->val[coord] >= curr->lo->val[coord])
	{
	  curr->lo->prev[coord] = current->prev[coord];
	  curr->lo->next[coord] = current;
	  if (current->prev[coord] == NULL)
	    obj->elist[coord] = curr->lo;
	  else
	    current->prev[coord]->next[coord] = curr->lo;
	  
	  current->prev[coord] = curr->lo;
	}
      else
	{
	  curr->lo->prev[coord] = current;
	  curr->lo->next[coord] = NULL;
	  current->next[coord] = curr->lo;
	}
      
    }
  
}



void deleteObjects(int id, NBody * obj) //deleting an AABB with given id.
{
  if (id >= obj->size)
    {
      //cerr<<"Should not get here since VCollide should send only valid ids\n";
      return;
    }
  
  if (obj->AABB_arr[id] == NULL)
    {
      //cerr<<"Should not get here since VCollide should send only valid ids\n";
      return;
    }
  
  AABB *curr = obj->AABB_arr[id];  //this is the AABB to be deleted.
  obj->AABB_arr[id] = NULL;        //remove it from the AABB array.
  
  //first, we delete all the three intervals from the corresponding lists.
  int coord;
  for (coord=0; coord<3; coord++)
    {
      //first delete the "lo" endpoint of the interval.
      if (curr->lo->prev[coord] == NULL)
	obj->elist[coord] = curr->lo->next[coord];
      else
	curr->lo->prev[coord]->next[coord] = curr->lo->next[coord];
      
      curr->lo->next[coord]->prev[coord] = curr->lo->prev[coord];
      
      //then, delete the "hi" endpoint.
      if (curr->hi->prev[coord] == NULL)
	obj->elist[coord] = curr->hi->next[coord];
      else
	curr->hi->prev[coord]->next[coord] = curr->hi->next[coord];
      
      if (curr->hi->next[coord] != NULL)
	curr->hi->next[coord]->prev[coord] = curr->hi->prev[coord];
      
    }
  
  //delete all entries involving this id from the set of 
  //overlapping pairs.
  obj->overlapping_pairs.DelPairsInvolvingId(id);
  
  //de-allocate the memory
  delete curr->lo;
  delete curr->hi;
  delete curr;
}




VCInternal::VCInternal()
{
  state = VCstate_default;
  next_id = 0;
  
  size = DEFAULT_SIZE;                   //set the size of the array.
  vc_objects = new VCObject*[size]; //allocate the array.
  int i;
  for (i=0; i<size; i++)
    vc_objects[i] = NULL;
  
  disabled.Clear();  //to begin with, no pairs are disabled.
}


VCInternal::~VCInternal()
{

  //deallocate the memory.
  int i;
  for (i=0; i<size; i++)
    {
      if (vc_objects[i])
	{
	  delete vc_objects[i]->b;
	  delete vc_objects[i];
	}
    }
  delete [] vc_objects;
}


//1. check if the size fit in
//2 assign the object an id and activate the object
int VCInternal::NewObject(int *id) //create a new object in the database.
{

  //increase the size of the "vc_objects" array if required.
  if (next_id >= size) 
    {
      int newsize = (next_id >= 2*size) ? (next_id+1) : 2*size;
      VCObject **temp = new VCObject*[newsize];
      int i;
      for (i=0; i<size; i++)
	temp[i] = vc_objects[i];
      for (i=size; i<newsize; i++)
	temp[i] = NULL;
      delete [] vc_objects;
      vc_objects = temp;
      size = newsize;
      
    }
  
  //allocate a new object.
  vc_objects[next_id] = new VCObject;
  
  *id = next_id;  //for returning the id generated by VCollide.
  current_id = next_id;
  vc_objects[next_id]->id = next_id;
  vc_objects[next_id]->b = new Object;
  vc_objects[next_id]->b->BeginModel();
  //_state = 1;//default the object is activate
  next_id++; 
  
  return 0;
}

int VCInternal::AddTri(double v1[], double v2[], double v3[]) 
{                     

  vc_objects[current_id]->b->AddTri(v1, v2, v3, 0);  //add triangle.
  return 0;
}

// 1. add current object to n body
// 2. have RAPID build the OBB tree.
// 3. initialize trans
int VCInternal::EndObject(void)
{  

  AddObject(current_id, vc_objects[current_id]->b, &nbody);
  
  vc_objects[current_id]->b->EndModel();

  memset( ( (void *)vc_objects[current_id]->trans), 0, 16*sizeof(double) );
  vc_objects[current_id]->trans[0][0] = 1.0;
  vc_objects[current_id]->trans[1][1] = 1.0;
  vc_objects[current_id]->trans[2][2] = 1.0;
  vc_objects[current_id]->trans[3][3] = 1.0;
  
  return 0;
  
}


int VCInternal::UpdateTrans(int id, double t[][4])
{           

  VCObject *current = vc_objects[id];
  
  //update the private copy of the transformation matrix.
  memcpy((void *)current->trans, (void *)t, 16*sizeof(double));
  
  //have the nbody database update itself appropriately.
  //updateteTrans(current->id, t, &nbody);
  updatetempTrans(current->id, t, &nbody);
  

  return 0;
  
}



int VCInternal::ActivatePair(int id1, int id2)
{
      disabled.DelPair(id1, id2);
      return 0;
}

int VCInternal::DeactivatePair(int id1, int id2)
{

      if (id1!=id2)
	disabled.AddPair(id1, id2);
      
    return 0;
}


int VCInternal::DeleteObject(int id) //delete an object from the database.
{


      delete vc_objects[id]->b; //delete the RAPID box.
      delete vc_objects[id];    //delete the object.
      vc_objects[id] = NULL; 
      
      disabled.DelPairsInvolvingId(id);

      deleteObjects(id, &nbody); //delete the object from the nbody database.
      return 0;

  
}



int VCInternal::Collide(void)  //perform collision detection.
{

  
  //Clear the results from earlier collision tests.
  report_data.Clear();
  
  //Simultaneously traverse the "overlapping_pairs" database and the 
  //"disabled_pairs" database and make calls to the RAPID collision
  //detection routine where required.
  int i;
   //std::cout<< nbody.overlapping_pairs.size<< std::endl;
  for (i=0; i<nbody.overlapping_pairs.size; i++)
    {
      Elem *curr_ovrlp = nbody.overlapping_pairs.arr[i];
      
      Elem *curr_disabled = i<disabled.size ? disabled.arr[i]: NULL;
      
      
	  while (curr_ovrlp != NULL)
		{

		  while (curr_disabled && curr_disabled->id <= curr_ovrlp->id)
			      curr_disabled = curr_disabled->next;

      double R1[3][3], T1[3], R2[3][3], T2[3];
      for (int index = 0; index < 9; index++){
        int x = index/3;
        int y = index%3;
        R1[x][y] = vc_objects[i]->trans[x][y];
        R2[x][y] = vc_objects[curr_ovrlp->id]->trans[x][y];
      }

      for (int index = 0; index < 3; index++){
        T1[index] = vc_objects[i]->trans[index][3];
        T2[index] = vc_objects[curr_ovrlp->id]->trans[index][3];
      }

			      //call the RAPID collision detection routine.
			::Collide(R1, T1, vc_objects[i]->b, R2, T2, vc_objects[curr_ovrlp->id]->b, FIRST_CONTACT);
			      
			      //if there is a collision, then add the pair to the
			      //collision report database.
			if (Object_num_contacts != 0)
			      report_data.AddPair(i, vc_objects[curr_ovrlp->id]->id);
			      
		  curr_ovrlp = curr_ovrlp->next;
		}
    }
  return 0;
}


//report the results of collision detection.
//sz is the size of the array pointed to by vcrep. If sz is less than
//the number of collision pairs, then fill the array with first sz number
//of collision pairs.
//Returns the total number of collision pairs.
int VCInternal::Report(int sz, VCReportType *vcrep)
{
  int no_of_colliding_pairs=0;
  int vc_rep_count = 0;
  
  int i;
  for (i=0; i<report_data.size; i++)
    {
      Elem *current;
      for (current=report_data.arr[i]; current != NULL; current=current->next)
	{
	  no_of_colliding_pairs++;
	  if (vc_rep_count <sz) //if the array is not full yet, then 
	    {                   //fill the data in it.
	      vcrep[vc_rep_count].id1 = i;
	      vcrep[vc_rep_count].id2 = current->id;
	      vc_rep_count++;
	    }
	}
      
    }
  return no_of_colliding_pairs;
}


