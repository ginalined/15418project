#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "CycleTimer.h"

extern float toBW(int bytes, float sec);
__global__ void
saxpy_kernel(int N, float alpha, float* x, float* y, float* result) {

    // compute overall index from position of thread in current block,
    // and given the block we are in
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N)
       result[index] = alpha * x[index] + y[index];
}

void
saxpyCuda(int N, float alpha, float* xarray, float* yarray, float* resultarray) {

    int totalBytes = sizeof(float) * 3 * N;

    // compute number of blocks and threads per block
    const int threadsPerBlock = 512;
    const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    float* device_x;
    float* device_y;
    float* device_result;

    //
    // allocate device memory buffers on the GPU using hipMalloc
    //
    hipMalloc(&device_x, N * sizeof(float));
    hipMalloc(&device_y, N * sizeof(float));
    hipMalloc(&device_result, N * sizeof(float));


    // start timing after allocation of device memory
    double startTime = CycleTimer::currentSeconds();

    //
    // copy input arrays to the GPU using hipMemcpy
    //
    double kernelCopyStartTime = CycleTimer::currentSeconds();
    hipMemcpy(device_x, xarray, N, hipMemcpyHostToDevice);
    hipMemcpy(device_y, yarray, N, hipMemcpyHostToDevice);
    double kernelCopyEndTime = CycleTimer::currentSeconds();


    // run kernel
    double kernelStartTime = CycleTimer::currentSeconds();
    saxpy_kernel<<<blocks, threadsPerBlock>>>(N, alpha, device_x, device_y, device_result);
    hipDeviceSynchronize();
    double kernelEndTime = CycleTimer::currentSeconds();

    //
    // copy result from GPU using hipMemcpy
    //
    hipMemcpy(resultarray, device_result, N, hipMemcpyDeviceToHost);

    // end timing after result has been copied back into host memory
    double endTime = CycleTimer::currentSeconds();

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    double overallDuration = endTime - startTime;
    double kernelRuntime = kernelEndTime - kernelStartTime;
    printf("Overall: %.3f ms\t\t[%.3f GB/s]\nKernel:%.3f ms\t\t[%.3f GB/s]\n\n", 
            1000.f * overallDuration, toBW(totalBytes, overallDuration), 
            1000.f * kernelRuntime, toBW(totalBytes, kernelRuntime));

    // free memory buffers on the GPU
    hipFree(device_x);
    hipFree(device_y);
    hipFree(device_result);

}